#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------
// This implementation is mainly from 
// 'https://github.com/Ignotus/theano-roi-align/blob/master/roi_align.cu'

#include <cfloat>
#include <stdio.h>
#include <math.h>
#include <float.h>

//#include "caffe/fast_rcnn_layers.hpp"
#include "caffe/roi_alignment2_layer.hpp"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
//__global__ void ROIAlignForward(const int nthreads, const Dtype* bottom_data,
//    const Dtype spatial_scale, const int channels, const int height,
//    const int width, const int pooled_height, const int pooled_width,
//    const Dtype* bottom_rois, Dtype* top_data, int* argmax_data)

__global__ void ROIAlignForward(const int nthreads, const Dtype* bottom_data,
	    const Dtype spatial_scale, const int channels, const int height,
	    const int width, const int pooled_height, const int pooled_width,
	    const Dtype* bottom_rois, Dtype* top_data,
	    Dtype* argmax_data_x, Dtype* argmax_data_y)
{
	  CUDA_KERNEL_LOOP(index, nthreads)
	{
	    // (n, c, ph, pw) is an element in the pooled output
	    int pw = index % pooled_width;
	    int ph = (index / pooled_width) % pooled_height;
	    int c = (index / pooled_width / pooled_height) % channels;
	    int n = index / pooled_width / pooled_height / channels;

	    bottom_rois += n * 5;
	    int roi_batch_ind = bottom_rois[0];
	    Dtype roi_start_w = bottom_rois[1] * spatial_scale;
	    Dtype roi_start_h = bottom_rois[2] * spatial_scale;
	    Dtype roi_end_w = bottom_rois[3] * spatial_scale;
	    Dtype roi_end_h = bottom_rois[4] * spatial_scale;

	    Dtype roi_width = roi_end_w - roi_start_w + 1;
	    Dtype roi_height = roi_end_h - roi_start_h + 1;

	    Dtype bin_size_h = static_cast<Dtype>(roi_height)
	                       / static_cast<Dtype>(pooled_height);
	    Dtype bin_size_w = static_cast<Dtype>(roi_width)
	                       / static_cast<Dtype>(pooled_width);

	    Dtype hstart = static_cast<Dtype>(ph) * bin_size_h;
	    Dtype wstart = static_cast<Dtype>(pw) * bin_size_w;
	    Dtype hend = static_cast<Dtype>(ph + 1) * bin_size_h;
	    Dtype wend = static_cast<Dtype>(pw + 1) * bin_size_w;

	    // Add roi offsets and clip to input boundaries
	    hstart = fminf(fmaxf(hstart + roi_start_h, 0.), static_cast<float>(height));
	    hend = fminf(fmaxf(hend + roi_start_h, 0.), static_cast<float>(height));
	    wstart = fminf(fmaxf(wstart + roi_start_w, 0.), static_cast<float>(width));
	    wend = fminf(fmaxf(wend + roi_start_w, 0.), static_cast<float>(width));
	    bool is_empty = (hend <= hstart) || (wend <= wstart);

	    // Define an empty pooling region to be zero
	    Dtype maxval = is_empty ? 0 : -FLT_MAX;
	    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
	    Dtype maxidx_x = -1;
	    Dtype maxidx_y = -1;
	    bottom_data += (roi_batch_ind * channels + c) * height * width;
	    for (Dtype h = hstart; h < hend; h += 1.)
	    {
	      for (Dtype w = wstart; w < wend; w += 1.)
	      {
	        // Selecting four regular locations for bilinear interpolation
	        int x_left = floor(w);
	        int x_right = ceil(w);
	        int y_bottom = floor(h);
	        int y_top = ceil(h);

	        int top_left_index = y_top * width + x_left;
	        int top_right_index = y_top * width + x_right;
	        int bottom_left_index = y_bottom * width + x_left;
	        int bottom_right_index = y_bottom * width + x_right;

	        bool is_top_left_in = x_left >= 0 && x_left <= width - 1 && y_top >= 0 && y_top <= height - 1;
	        bool is_top_right_in = x_right >= 0 && x_right <= width - 1 && y_top >= 0 && y_top <= height - 1;
	        bool is_bottom_left_in = x_left >= 0 && x_left <= width - 1 && y_bottom >= 0 && y_bottom <= height - 1;
	        bool is_bottom_right_in = x_right >= 0 && x_right <= width - 1 && y_bottom >= 0 && y_bottom <= height - 1;

	        Dtype val = 0;
	        if (is_top_left_in)
	          val += (1 - w + x_left) * (1 - y_top + h) * bottom_data[top_left_index];
	        if (is_top_right_in)
	          val += (1 - x_right + w) * (1 - y_top + h) * bottom_data[top_right_index];
	        if (is_bottom_left_in)
	          val += (1 - w + x_left) * (1 - h + y_bottom) * bottom_data[bottom_left_index];
	        if (is_bottom_right_in)
	          val += (1 - x_right + w) * (1 - h + y_bottom) * bottom_data[bottom_right_index];

	        if (val > maxval)
	        {
	          maxval = val;
	          maxidx_x = w; // float number
	          maxidx_y = h; // float number
	        }
	      }
	    }
	    top_data[index] = maxval;
	    argmax_data_x[index] = maxidx_x;
	    argmax_data_y[index] = maxidx_y;
	}
}

template <typename Dtype>
void ROIAlignment2Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
{
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
//  int* argmax_data = max_idx_.mutable_gpu_data();
  /////////////////////////////////////////////////
  Dtype* argmax_data_x = max_idx_x.mutable_gpu_data();
  Dtype* argmax_data_y = max_idx_y.mutable_gpu_data();
  /////////////////////////////////////////////////
  int count = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
//  ROIAlignForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
//      count, bottom_data, spatial_scale_, channels_, height_, width_,
//      pooled_height_, pooled_width_, bottom_rois, top_data, argmax_data);
  ROIAlignForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, spatial_scale_, channels_, height_, width_,
      pooled_height_, pooled_width_, bottom_rois, top_data,
      argmax_data_x, argmax_data_y);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
//__global__ void ROIAlignBackward(const int nthreads, const Dtype* top_diff,
//    const int* argmax_data, const int num_rois, const Dtype spatial_scale,
//    const int channels, const int height, const int width,
//    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
//    const Dtype* bottom_rois)
__global__ void ROIAlignBackward(const int nthreads, const Dtype* top_diff, const Dtype* argmax_data_x,
	    const Dtype* argmax_data_y, const int num_rois, const Dtype spatial_scale,
	    const int channels, const int height, const int width,
	    const int pooled_height, const int pooled_width, Dtype* bottom_diff,
	    const Dtype* bottom_rois)
{
	 CUDA_KERNEL_LOOP(index, nthreads) {
	    // (n, c, h, w) coords in bottom data
	    int w = index % width;
	    int h = (index / width) % height;
	    int c = (index / width / height) % channels;
	    int n = index / width / height / channels;

	    Dtype gradient = 0;
	    // Accumulate gradient over all ROIs that pooled this element
	    for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
	      const Dtype* offset_bottom_rois = bottom_rois + roi_n * 5;
	      int roi_batch_ind = offset_bottom_rois[0];
	      // Skip if ROI's batch index doesn't match n
	      if (n != roi_batch_ind) {
	        continue;
	      }

	      // And it assumes that we don't have any negative offset of course
	      int roi_start_w = floor(offset_bottom_rois[1] * spatial_scale);
	      int roi_start_h = floor(offset_bottom_rois[2] * spatial_scale);
	      int roi_end_w = ceil(offset_bottom_rois[3] * spatial_scale);
	      int roi_end_h = ceil(offset_bottom_rois[4] * spatial_scale);

	      // Skip if ROI doesn't include (h, w)
//	      const bool in_roi = (w >= roi_start_w && w <= roi_end_w &&
//	                           h >= roi_start_h && h <= roi_end_h);
	      const bool in_roi = (w >= roi_start_w-1 && w <= roi_end_w+1 && h >= roi_start_h-1 && h <= roi_end_h+1);

	      if (!in_roi) {
	        continue;
	      }

	      int offset = (roi_n * channels + c) * pooled_height * pooled_width;
	      const Dtype* offset_top_diff = top_diff + offset;
	      const Dtype* offset_argmax_data_x = argmax_data_x + offset;
	      const Dtype* offset_argmax_data_y = argmax_data_y + offset;

	      // Compute feasible set of pooled units that could have pooled
	      // this bottom unit
	      Dtype roi_width = roi_end_w - roi_start_w + 1;
	      Dtype roi_height = roi_end_h - roi_start_h + 1;

	      Dtype bin_size_h = static_cast<Dtype>(roi_height)
	                         / static_cast<Dtype>(pooled_height);
	      Dtype bin_size_w = static_cast<Dtype>(roi_width)
	                         / static_cast<Dtype>(pooled_width);

	      int phstart = floor(static_cast<Dtype>(h - roi_start_h) / bin_size_h);
	      int phend = ceil(static_cast<Dtype>(h - roi_start_h + 1) / bin_size_h);
	      int pwstart = floor(static_cast<Dtype>(w - roi_start_w) / bin_size_w);
	      int pwend = ceil(static_cast<Dtype>(w - roi_start_w + 1) / bin_size_w);

	      phstart = min(max(phstart, 0), pooled_height);
	      phend = min(max(phend, 0), pooled_height);
	      pwstart = min(max(pwstart, 0), pooled_width);
	      pwend = min(max(pwend, 0), pooled_width);

	      phstart = 0;
	      phend = pooled_height;
	      pwstart = 0;
	      pwend = pooled_width;

	      for (int ph = phstart; ph < phend; ++ph) {
	        for (int pw = pwstart; pw < pwend; ++pw) {
	          int index = ph * pooled_width + pw;
	          Dtype max_x = offset_argmax_data_x[index];
	          Dtype max_y = offset_argmax_data_y[index];

	          int x_left = floor(max_x);
	          int x_right = ceil(max_x);
	          int y_bottom = floor(max_y);
	          int y_top = ceil(max_y);

	          if (x_left == w && y_top == h)
	            gradient += (1 - max_x + x_left) * (1 - y_top + max_y) * offset_top_diff[index];
	          if (x_left == w && y_bottom == h)
	            gradient += (1 - max_x + x_left) * (1 - max_y + y_bottom)  * offset_top_diff[index];
	          if (x_right == w && y_top == h)
	            gradient += (1 - x_right + max_x) * (1 - y_top + max_y) * offset_top_diff[index];
	          if (x_right == w && y_bottom == h)
	            gradient += (1 - x_right + max_x) * (1 - max_y + y_bottom) * offset_top_diff[index];
	        }
	      }
	    }
	    bottom_diff[index] = gradient;
	}
}

template <typename Dtype>
void ROIAlignment2Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
//  const int* argmax_data = max_idx_.gpu_data();
  /////////////////////////////////////////////////////////////////
  const Dtype* argmax_data_x = max_idx_x.gpu_data();
  const Dtype* argmax_data_y = max_idx_y.gpu_data();
  ////////////////////////////////////////////////////////////////
  // NOLINT_NEXT_LINE(whitespace/operators)
//  ROIAlignBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
//      count, top_diff, argmax_data, top[0]->num(), spatial_scale_, channels_,
//      height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
  ROIAlignBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff,
      argmax_data_x, argmax_data_y,
      top[0]->num(), spatial_scale_, channels_,
      height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignment2Layer);

}  // namespace caffe
